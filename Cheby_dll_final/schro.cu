#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<>
#include<stdint.h>
#include<stdio.h>

#define gram(__ptr,__ty,__n) hipMalloc((void**)&__ptr,(__n)*sizeof(__ty))
#define eidx(__i,__j) ((__i)*H+(__j))*2
#define oidx(__i,__j) eidx(__i,__j)+1
#define idx(__i,__j) (__i)*H*2+(__j) //δ����2��������ż��
#include"schro.cuh"
#include"lazy_cheby_exp.h"

//�ȷ��̲���ͨ��
__global__ void global_five_point_heat(real* src, real* dst, real a) {
	iint i = blockIdx.x;
	iint j = threadIdx.x;
	bool odd = j % 2;
	j /= 2;
	if (odd) { //�鲿
		if (i == 0 || i == W - 1 || j == 0 || j == H - 1) {
			dst[oidx(i, j)] = 0;
		}
		else {
			dst[oidx(i, j)] = (1 - 4 * a) * src[oidx(i, j)] //����λ--�鲿
				+ a * (src[oidx(i, j - 1)] + src[oidx(i, j + 1)]
					+ src[oidx(i - 1, j)] + src[oidx(i + 1, j)]);
		}
	}
	else { //ʵ��
		if (i == 0 || i == W - 1 || j == 0 || j == H - 1) {
			dst[eidx(i, j)] = 0;
		}
		else {
			dst[eidx(i, j)] = (1 - 4 * a) * src[eidx(i, j)] //ż��λ--ʵ��
				+ a * (src[eidx(i, j - 1)] + src[eidx(i, j + 1)]
					+ src[eidx(i - 1, j)] + src[eidx(i + 1, j)]);
		}
	}
}

__global__ void global_add(real* dst_a, real* b) {
	iint i = blockIdx.x;
	iint j = threadIdx.x;
	dst_a[idx(i, j)] += b[idx(i, j)];
}
__global__ void global_minus(real* dst_a, real* b) {
	iint i = blockIdx.x;
	iint j = threadIdx.x;
	dst_a[idx(i, j)] -= b[idx(i, j)];
}

#define re eidx(i,j)
#define im oidx(i,j)

__global__ void global_add_i(real* dst_a, real* b) {
	iint i = blockIdx.x;
	iint j = threadIdx.x;
	bool odd = j % 2;
	j /= 2;
	if (odd) {
		dst_a[im] += b[re];
	}
	else {
		dst_a[re] -= b[im];
	}
}
__global__ void global_minus_i(real* dst_a, real* b) {
	iint i = blockIdx.x;
	iint j = threadIdx.x;
	bool odd = j % 2;
	j /= 2;
	if (odd) {
		dst_a[im] -= b[re];
	}
	else {
		dst_a[re] += b[im];
	}
}
__global__ void global_mulnum(real* dst_a, real coef) {
	iint i = blockIdx.x;
	iint j = threadIdx.x;
	dst_a[idx(i, j)] *= coef;
}

inline void gpu_add(real* dst_a, real* b) {
	global_add << <W, 2 * H >> > (dst_a, b);
}
inline void gpu_minus(real* dst_a, real* b) {
	global_minus << <W, 2 * H >> > (dst_a, b);
}
inline void gpu_add_i(real* dst_a, real* b) {
	global_add_i << <W, 2 * H >> > (dst_a, b);
}
inline void gpu_minus_i(real* dst_a, real* b) {
	global_minus_i << <W, 2 * H >> > (dst_a, b);
}
inline void gpu_mulnum(real* dst_a, real coef) {
	global_mulnum << <W, 2 * H >> > (dst_a, coef);
}
//�˽ӿڱ������
/*
inline void gpu_init(real* obj) {
	gram(obj, real, W * H * 2);
}*/
////////////////
inline void gpu_free(real* obj) {
	hipFree(obj);
}
inline void gpu_copy(real* src, real* dst) {
	hipMemcpy(dst, src, DSIZE, hipMemcpyDeviceToDevice);
}
inline void gpu_output(real* src, real* dst) {
	hipMemcpy(dst, src, DSIZE, hipMemcpyDeviceToHost);
}
inline void gpu_upload(real* src, real* dst) {
	hipMemcpy(dst, src, DSIZE, hipMemcpyHostToDevice);
}

//����ַ�����ͨ��
__global__ void global_five_point_legacy(real* src, real* dst, real a) {
	iint i = blockIdx.x;
	iint j = threadIdx.x;
	bool odd = j % 2;
	j /= 2;
	if (odd) { //�鲿
		if (i == 0 || i == W - 1 || j == 0 || j == H - 1) {
			dst[oidx(i, j)] = 0;
		}
		else { 
			dst[im] = src[im] - 4 * a * src[re] +
				a * (src[eidx(i - 1, j)] + src[eidx(i + 1, j)] +
					src[eidx(i, j - 1)] + src[eidx(i, j + 1)]);
		}
	}
	else { //ʵ��
		if (i == 0 || i == W - 1 || j == 0 || j == H - 1) {
			dst[eidx(i, j)] = 0;
		}
		else {
			dst[re] = src[re] + 4 * a * src[im] -
				a * (src[oidx(i - 1, j)] + src[oidx(i + 1, j)] +
					src[oidx(i, j - 1)] + src[oidx(i, j + 1)]);
		}
	}
}

__global__ void global_five_point(real* src, real* dst, real a) {
	iint i = blockIdx.x;
	iint j = threadIdx.x;
	bool odd = j % 2;
	j /= 2;
	if (odd) { //�鲿
		if (i == 0 || i == W - 1 || j == 0 || j == H - 1) {
			dst[im] = 0;
		}
		else {
			dst[im] = a * (src[oidx(i - 1, j)] + src[oidx(i + 1, j)] +
				src[oidx(i, j - 1)] + src[oidx(i, j + 1)] - 4 * src[im]);
		}
	}
	else { //ʵ��
		if (i == 0 || i == W - 1 || j == 0 || j == H - 1) {
			dst[re] = 0;
		}
		else {
			dst[re] = a * (src[eidx(i - 1, j)] + src[eidx(i + 1, j)] +
				src[eidx(i, j - 1)] + src[eidx(i, j + 1)] - 4 * src[re]);
		}
	}
}

inline void gpu_five_point(real* src, real* dst, real a) {
	global_five_point << <W, 2 * H >> > (src, dst, a);
}
inline void naive_five_point(real* src, real* dst, real a) {
	global_five_point_legacy << <W, 2 * H >> > (src, dst, a);
}

inline void host_naive_five_point(real* src, real* dst, real a) {
	for (int n = 0; n < W * H * 2; n++) {
		iint i = n / (H * 2);
		iint j = n % (H * 2);
		bool odd = j % 2;
		j /= 2;
		if (odd) { //�鲿
			if (i == 0 || i == W - 1 || j == 0 || j == H - 1) {
				dst[oidx(i, j)] = 0;
			}
			else {
				dst[im] = src[im] - 4 * a * src[re] +
					a * (src[eidx(i - 1, j)] + src[eidx(i + 1, j)] +
						src[eidx(i, j - 1)] + src[eidx(i, j + 1)]);
			}
		}
		else { //ʵ��
			if (i == 0 || i == W - 1 || j == 0 || j == H - 1) {
				dst[eidx(i, j)] = 0;
			}
			else {
				dst[re] = src[re] + 4 * a * src[im] -
					a * (src[oidx(i - 1, j)] + src[oidx(i + 1, j)] +
						src[oidx(i, j - 1)] + src[oidx(i, j + 1)]);
			}
		}
	}
}

inline void host_five_point(real* src, real* dst, real a) {
	int i, j; bool odd;
	for (int n = 0; n < W * H * 2; n++) {
		i = n / (H * 2);
		j = n % (H * 2);
		odd = j % 2;
		j /= 2;
		if (odd) { //�鲿
			if (i == 0 || i == W - 1 || j == 0 || j == H - 1) {
				dst[im] = 0;
			}
			else {
				dst[im] = a * (src[oidx(i - 1, j)] + src[oidx(i + 1, j)] +
					src[oidx(i, j - 1)] + src[oidx(i, j + 1)] - 4 * src[im]);
			}
		}
		else { //ʵ��
			if (i == 0 || i == W - 1 || j == 0 || j == H - 1) {
				dst[re] = 0;
			}
			else {
				dst[re] = a * (src[eidx(i - 1, j)] + src[eidx(i + 1, j)] +
					src[eidx(i, j - 1)] + src[eidx(i, j + 1)] - 4 * src[re]);
			}
		}
	}
}

#define loop(__i,__n) for(int __i=0;__i<__n;__i++)
#define WH2 W*H*2
inline void host_add(real* dst_a, real* b) {
	loop(i, WH2) {
		dst_a[i] += b[i];
	}
}
inline void host_minus(real* dst_a, real* b) {
	loop(i, WH2) {
		dst_a[i] -= b[i];
	}
}
inline void host_add_i(real* dst_a, real* b) {
	bool odd; int i, j;
	loop(n, WH2) {
		i = n / (H * 2);
		j = n % (H * 2);
		odd = j % 2;
		j /= 2;
		if (odd) {
			dst_a[im] += b[re];
		}
		else {
			dst_a[re] -= b[im];
		}
	}
}
inline void host_minus_i(real* dst_a, real* b) {
	bool odd; int i, j;
	loop(n, WH2) {
		i = n / (H * 2);
		j = n % (H * 2);
		odd = j % 2;
		j /= 2;
		if (odd) {
			dst_a[im] -= b[re];
		}
		else {
			dst_a[re] += b[im];
		}
	}
}
inline void host_mulnum(real* dst_a, real coef) {
	loop(i, WH2) {
		dst_a[i] *= coef;
	}
}

__global__ void global_wall(real* dst, int* wall) {
	iint i = blockIdx.x;
	iint j = threadIdx.x;
	if (wall[i * H + j / 2]) {
		dst[idx(i, j)] = 0;
	}
}
inline void gpu_wall(real* dst, int* wall) {
	global_wall << <W, 2 * H >> > (dst, wall);
}
inline void host_wall(real* dst, int* wall) {
	iint i, j;
	loop(n, WH2) {
		i = n / H;
		j = n % H;
		j /= 2;
		if (wall[i * H + j]) {
			dst[eidx(i, j)] = 0;
			dst[eidx(i, j)] = 0;
		}
	}
}